#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32        
#define MAX_THREADS 1024   
#define SHARED_MEM_SIZE 128 * 1024  
#define CONSTANT_MEM_SIZE 64 * 1024 


__constant__ int constant_NNZ_data[];
__constant__ int constant_rowind[];
__constant__ float constant_sparsity_list[];


__global__ void ApSpGEMMKernel(int* d_A, int* d_B, int* d_C, int numRows, int numCols, int numNNZ) {

    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalIdx < numRows) {

    }
}


int main() {

    int *d_A, *d_B, *d_C;
    int numRows;
    int numCols ;
    int numNNZ ;


    hipMalloc((void**)&d_A, numNNZ * sizeof(int));
    hipMalloc((void**)&d_B, numCols * sizeof(int));
    hipMalloc((void**)&d_C, numRows * sizeof(int));


    hipMemcpyToSymbol(HIP_SYMBOL(constant_NNZ_data), /* data */, sizeof(int) * numNNZ);
    hipMemcpyToSymbol(HIP_SYMBOL(constant_rowind), /* data */, sizeof(int) * numRows);
    hipMemcpyToSymbol(HIP_SYMBOL(constant_sparsity_list), /* data */, sizeof(float) * numRows);


    int numBlocks = (numRows + MAX_THREADS - 1) / MAX_THREADS;
    ApSpGEMMKernel<<<numBlocks, MAX_THREADS>>>(d_A, d_B, d_C, numRows, numCols, numNNZ);
    

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // hipMemcpy(C, d_C, numRows * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
